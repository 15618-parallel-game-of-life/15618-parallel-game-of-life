#include "hip/hip_runtime.h"
#include <string>
#include <algorithm>
#define _USE_MATH_DEFINES
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaRenderer.h"
#include "image.h"

#define SQRT_NUM_THREADS 16

#define DEBUG

#ifdef DEBUG
#define cudaCheckError(ans) cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort = true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "CUDA Error: %s at %s:%d\n",
            hipGetErrorString(code), file, line);
    if (abort)
      exit(code);
  }
}
#else
#define cudaCheckError(ans) ans
#endif

// data structure definition start here

// global constants definition
struct GlobalConstants
{
  int size;
  int pixelSize;
  float *imageData;
};

__constant__ GlobalConstants cuConstRendererParams;

// data structure definition end here

// CUDA kernel functions start here

__global__ void kernelAdvanceAnimationWithBuffer(uint8_t *current, uint8_t *next)
{
  __shared__ uint8_t buffer[SQRT_NUM_THREADS + 2][SQRT_NUM_THREADS + 2];

  int idxX = blockIdx.x * blockDim.x + threadIdx.x;
  int idxY = blockIdx.y * blockDim.y + threadIdx.y;

  int size = cuConstRendererParams.size;

  // load the neighborhood elements into local buffer
  int bufferX = threadIdx.x + 1;
  int bufferY = threadIdx.y + 1;
  int globalIdx = idxX * size + idxY;

  if (idxX < size && idxY < size)
  {
    buffer[bufferX][bufferY] = current[globalIdx];

    // boundaries
    if (threadIdx.x == 0)
      buffer[0][bufferY] = current[((idxX - 1 + size) % size) * size + idxY];
    if (threadIdx.x == blockDim.x - 1)
      buffer[SQRT_NUM_THREADS + 1][bufferY] = current[((idxX + 1) % size) * size + idxY];
    if (threadIdx.y == 0)
      buffer[bufferX][0] = current[idxX * size + ((idxY - 1 + size) % size)];
    if (threadIdx.y == blockDim.y - 1)
      buffer[bufferX][SQRT_NUM_THREADS + 1] = current[idxX * size + ((idxY + 1) % size)];
    if (threadIdx.x == 0 && threadIdx.y == 0)
      buffer[0][0] = current[((idxX - 1 + size) % size) * size + ((idxY - 1 + size) % size)];
    if (threadIdx.x == 0 && threadIdx.y == blockDim.y - 1)
      buffer[0][SQRT_NUM_THREADS + 1] = current[((idxX - 1 + size) % size) * size + ((idxY + 1) % size)];
    if (threadIdx.x == blockDim.x - 1 && threadIdx.y == 0)
      buffer[SQRT_NUM_THREADS + 1][0] = current[((idxX + 1) % size) * size + ((idxY - 1 + size) % size)];
    if (threadIdx.x == blockDim.x - 1 && threadIdx.y == blockDim.y - 1)
      buffer[SQRT_NUM_THREADS + 1][SQRT_NUM_THREADS + 1] = current[((idxX + 1) % size) * size + ((idxY + 1) % size)];
  }

  __syncthreads();

  if (idxX >= size || idxY >= size)
    return;

  int count = 0;
  for (int ii = -1; ii <= 1; ii++)
  {
    for (int jj = -1; jj <= 1; jj++)
    {
      if (buffer[bufferX + ii][bufferY + jj] && !(ii == 0 && jj == 0))
        count++;
    }
  }

  // update next state
  if (count == 3 || (count == 2 && buffer[bufferX][bufferY]))
    next[globalIdx] = 1;
  else
    next[globalIdx] = 0;
}

__global__ void kernelAdvanceAnimation(uint8_t *current, uint8_t *next)
{
  int idxX = blockIdx.x * blockDim.x + threadIdx.x;
  int idxY = blockIdx.y * blockDim.y + threadIdx.y;

  int size = cuConstRendererParams.size;

  if (idxX >= size || idxY >= size)
    return;

  int idx = (idxX * size + idxY);
  int count = 0;
  for (int ii = idxX - 1; ii <= idxX + 1; ii++)
  {
    for (int jj = idxY - 1; jj <= idxY + 1; jj++)
    {
      int idx2 = ((ii + size) % size) * size + (jj + size) % size;
      if (idx2 == idx)
        continue;
      if (current[idx2])
        count++;
    }
  }

  if (count == 3 || (count == 2 && current[idx]))
    next[idx] = 1;
  else
    next[idx] = 0;
}

__global__ void kernelRenderFrame(uint8_t *frame)
{
  int imageX = blockIdx.x * blockDim.x + threadIdx.x;
  int imageY = blockIdx.y * blockDim.y + threadIdx.y;

  int size = cuConstRendererParams.size;
  int pixelSize = cuConstRendererParams.pixelSize;

  if (imageX >= (size * pixelSize) || imageY >= (size * pixelSize))
    return;

  int idxX = imageX / pixelSize;
  int idxY = imageY / pixelSize;

  int idx = (idxX * size + idxY);
  uint8_t c = frame[idx];
  float4 imgPixel;
  if (c == 1)
    imgPixel = make_float4(1.0f, 1.0f, 1.0f, 1.0f);
  else
    imgPixel = make_float4(0.0f, 0.0f, 0.0f, 1.0f);

  int idx4 = 4 * (imageX * size * pixelSize + imageY);
  *(float4 *)(&cuConstRendererParams.imageData[idx4]) = imgPixel;
}

// CUDA kernel functions end here

// C++ class member functions start here

CudaRenderer::~CudaRenderer()
{
  if (image)
    delete image;
  if (deviceCurrentFrame)
  {
    hipFree(deviceCurrentFrame);
    hipFree(deviceNextFrame);
    hipFree(deviceImageData);
  }
  if (tmpFrame)
  {
    delete[] tmpFrame;
  }
}

const uint8_t *CudaRenderer::getFrame()
{
  hipMemcpy(tmpFrame,
             deviceCurrentFrame,
             sizeof(uint8_t) * size * size,
             hipMemcpyDeviceToHost);

  return tmpFrame;
}

const Image *CudaRenderer::getImage()
{
  hipMemcpy(image->data,
             deviceImageData,
             sizeof(float) * 4 * image->width * image->height,
             hipMemcpyDeviceToHost);

  return image;
}

void CudaRenderer::setup()
{
  int deviceCount = 0;
  std::string name;
  hipError_t err = hipGetDeviceCount(&deviceCount);

  // printf("---------------------------------------------------------\n");
  // printf("Initializing CUDA for CudaRenderer\n");
  // printf("Found %d CUDA devices\n", deviceCount);

  for (int i = 0; i < deviceCount; i++)
  {
    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, i);
    name = deviceProps.name;

    // printf("Device %d: %s\n", i, deviceProps.name);
    // printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
    // printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
    // printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
  }
  // printf("---------------------------------------------------------\n");

  // allocate memory storage on device
  hipMalloc(&deviceCurrentFrame, sizeof(uint8_t) * size * size);
  hipMalloc(&deviceNextFrame, sizeof(uint8_t) * size * size);
  hipMalloc(&deviceImageData, sizeof(float) * 4 * size * size * pixelSize * pixelSize);

  // copy data from input to device memory
  hipMemcpy(deviceCurrentFrame, initFrame, sizeof(uint8_t) * size * size, hipMemcpyHostToDevice);

  // initialize global constants with struct
  GlobalConstants params;
  params.size = size;
  params.pixelSize = pixelSize;
  params.imageData = deviceImageData;

  hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));
}

void CudaRenderer::advanceAnimation()
{
  dim3 blockDim(SQRT_NUM_THREADS, SQRT_NUM_THREADS, 1);
  dim3 gridDim(
      (image->width + blockDim.x - 1) / blockDim.x,
      (image->height + blockDim.y - 1) / blockDim.y);

  // pixel-level parallelism
  kernelAdvanceAnimationWithBuffer<<<gridDim, blockDim>>>(deviceCurrentFrame, deviceNextFrame);
  // kernelAdvanceAnimation<<<gridDim, blockDim>>>(deviceCurrentFrame, deviceNextFrame);
  cudaCheckError(hipDeviceSynchronize());

  // swap currentFrame and nextFrame
  std::swap(deviceCurrentFrame, deviceNextFrame);
}

void CudaRenderer::render()
{
  dim3 blockDim(SQRT_NUM_THREADS, SQRT_NUM_THREADS, 1);
  dim3 gridDim(
      (image->width + blockDim.x - 1) / blockDim.x,
      (image->height + blockDim.y - 1) / blockDim.y);

  // pixel-level parallelism
  kernelRenderFrame<<<gridDim, blockDim>>>(deviceCurrentFrame);
  cudaCheckError(hipDeviceSynchronize());
}

// C++ class member functions end here
